#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

/*
Total device time elapsed: 1.18128ms for 1024 * 1024 * 128 elements
Test PASSED
Done
*/

#define CHECK_CUDA_ERROR(call)                                                 \
    {                                                                          \
        const hipError_t error = call;                                        \
        if (error != hipSuccess) {                                            \
            std::cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", "      \
                      << "code: " << error << ", reason: "                     \
                      << hipGetErrorString(error) << std::endl;               \
            exit(1);                                                           \
        }                                                                      \
    }

// CUDA Kernel function to add elements of two arrays
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    // Size of vectors
    size_t numElements = 1024 * 1024 * 128;
    size_t size = numElements * sizeof(float);

    // Host vectors
    std::vector<float> h_A(numElements);
    std::vector<float> h_B(numElements);
    std::vector<float> h_C(numElements);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Device vectors
    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;

    // Allocate memory on host and device
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy input vectors from host memory to device memory
    hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements); //warmup

    // Start timer
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    CHECK_CUDA_ERROR(hipEventRecord(start, 0));

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Check for any errors launching the kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Failed to launch vectorAdd kernel (error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Stop timer
    CHECK_CUDA_ERROR(hipEventRecord(stop, 0));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));

    // Calculate elapsed time
    float elapsedTime;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    std::cout << "Total device time elapsed: "<<elapsedTime <<"ms \n";

    // Copy the device result vector back to the host result vector
    hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost);

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            std::cerr << "Result verification failed at element " << i << "!\n";
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "Test PASSED\n";

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    std::cout << "Done\n";
    return 0;
}